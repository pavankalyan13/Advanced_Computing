
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#define N (2048*2048)
#define N (2*2048)
#define THREADS_PER_BLOCK 512


__global__ void add(int *a, int *b, int *c)
 {
int index = threadIdx.x + blockIdx.x * blockDim.x;
//    printf(" index  = %d\n ", index);
    c[index] = a[index] + b[index];
}

void random_ints(int* x, int size)
{
        int i;
        for (i=0;i<size;i++) {
                x[i]=rand()%50;
        }
}

    int main(void) {
 int tempCounter =0;
 int *a, *b, *c; // host copies of a, b, c
 int *d_a, *d_b, *d_c; // device copies            
 int size = N * sizeof(int);
  // Alloc space for device copies of a, b, c
        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);
a = (int *)malloc(size); random_ints(a, N);
b = (int *)malloc(size); random_ints(b, N);
c = (int *)malloc(size);

 // Copy inputs to device
hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        // Launch add() kernel on GPU
add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
        // Copy result back to host
hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
       // Cleanup
       for(tempCounter =0; tempCounter < N; tempCounter++)
       {
        printf("%d + %d  = %d \n", a[tempCounter], b[tempCounter], c[tempCounter]);
       }
      
        free(a); free(b); free(c);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

